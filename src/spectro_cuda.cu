/**
 **  ______  ______  ___   ___  ______  ______  ______  ______
 ** /_____/\/_____/\/___/\/__/\/_____/\/_____/\/_____/\/_____/\
 ** \:::_ \ \::::_\/\::.\ \\ \ \:::_ \ \:::_ \ \::::_\/\:::_ \ \
 **  \:\ \ \ \:\/___/\:: \/_) \ \:\ \ \ \:\ \ \ \:\/___/\:(_) ) )_
 **   \:\ \ \ \::___\/\:. __  ( (\:\ \ \ \:\ \ \ \::___\/\: __ `\ \
 **    \:\/.:| \:\____/\: \ )  \ \\:\_\ \ \:\/.:| \:\____/\ \ `\ \ \
 **     \____/_/\_____\/\__\/\__\/ \_____\/\____/_/\_____\/\_\/ \_\/
 **
 **
 **   If you have downloaded the source code for "Dekoder for Morse" and are reading this,
 **   then thank you from the bottom of our hearts for making use of our hard work, sweat
 **   and tears in whatever you are implementing this into!
 **
 **   Copyright (C) 2020. GekkoFyre.
 **
 **   Dekoder for Morse is free software: you can redistribute it and/or modify
 **   it under the terms of the GNU General Public License as published by
 **   the Free Software Foundation, either version 3 of the License, or
 **   (at your option) any later version.
 **
 **   Dekoder is distributed in the hope that it will be useful,
 **   but WITHOUT ANY WARRANTY; without even the implied warranty of
 **   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 **   GNU General Public License for more details.
 **
 **   You should have received a copy of the GNU General Public License
 **   along with Dekoder for Morse.  If not, see <http://www.gnu.org/licenses/>.
 **
 **
 **   The latest source code updates can be obtained from [ 1 ] below at your
 **   discretion. A web-browser or the 'git' application may be required.
 **
 **   [ 1 ] - https://code.gekkofyre.io/phobos-dthorga/small-world-deluxe
 **
 ****************************************************************************************************/

#include "spectro_cuda.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>

/**
 * @brief SpectroFFTW::processCUDAFFT
 * @author Ville Räisänen <https://github.com/vsr83/QSpectrogram/blob/master/fftcuda.cu>
 * @param inputData
 * @param outputData
 * @param numSamples
 */
void processCUDAFFT(float *inputData, float *outputData, unsigned int numSamples)
{
    hipfftHandle plan;
    hipfftComplex *inputDataG, *outputDataG;
    int i;

    float *inputDataC, *outputDataC;
    outputDataC = (float *)malloc(sizeof(float) * numSamples * 2);
    inputDataC  = (float *)malloc(sizeof(float) * numSamples * 2);

    for (i = 0; i < numSamples; ++i) {
      inputDataC[i * 2]     = inputData[i];
      inputDataC[i * 2 + 1] = 0.0f;
    }

    hipMalloc((void **)&inputDataG,  sizeof(hipfftComplex) * numSamples);
    hipMalloc((void **)&outputDataG, sizeof(hipfftComplex) * numSamples);

    hipMemcpy(inputDataG, inputDataC, sizeof(hipfftComplex) * numSamples, hipMemcpyHostToDevice);
    hipfftPlan1d(&plan, numSamples, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, inputDataG, outputDataG, HIPFFT_FORWARD);
    hipfftDestroy(plan);
    hipMemcpy(outputDataC, outputDataG, sizeof(hipfftComplex) * numSamples, hipMemcpyDeviceToHost);
    hipFree(inputDataG);
    hipFree(outputDataG);

    for (i = 0; i < numSamples; ++i) {
        outputData[i] = outputDataC[i * 2];
    }

    free(outputDataC);
    free(inputDataC);

    return;
}
